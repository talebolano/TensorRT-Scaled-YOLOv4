#include "hip/hip_runtime.h"
#include "resize.h"
#include <hip/hip_runtime.h>
#include <>


__forceinline__ __device__ float3 get(uchar3* src, int x,int y,int w,int h){
    if(x < 0 || x>=w || y<0 || y>=h) return make_float3(0.,0.,0.);
    uchar3 temp = src[y*w + x];
    return make_float3(float(temp.x)/255., 
                        float(temp.y)/255.,
                        float(temp.z)/255);
}
__global__ void resizeNormKernel(uchar3* src,float *dst,int dstW, int dstH,int srcW,int srcH,
                                                float scaleX, float scaleY,float shiftX, float shiftY) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int x = idx % dstW;
    const int y = idx / dstW;
    if (x >= dstW || y >= dstH)
        return;
    float w = (x - shiftX + 0.5) * scaleX - 0.5;
    float h = (y - shiftY + 0.5) * scaleY - 0.5;
    int h_low = (int)h;
    int w_low = (int)w;
    int h_high = h_low + 1;
    int w_high = w_low + 1;
    float lh = h - h_low;
    float lw = w - w_low;
    float hh = 1 - lh, hw = 1 - lw;
    float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
    float3 v1 = get(src,w_low,h_low,srcW,srcH);
    float3 v2 = get(src,w_high,h_low,srcW,srcH);
    float3 v3 = get(src,w_low,h_high,srcW,srcH);
    float3 v4 = get(src,w_high,h_high,srcW,srcH);
    int stride = dstW*dstH;
    dst[y*dstW + x] = w1 *v1.x + w2 * v2.x + w3 *v3.x + w4 * v4.x ;
    dst[stride + y*dstW + x] = w1 *v1.y + w2 * v2.y + w3 *v3.y + w4 * v4.y ;
    dst[stride*2 + y*dstW + x] = w1 *v1.z + w2 * v2.z + w3 *v3.z + w4 * v4.z;
}

int resizeAndNorm(void * p,float *d,int w,int h,int in_w,int in_h, bool keepration ,bool keepcenter){
    float scaleX = (w*1.0f / in_w);
    float scaleY = (h*1.0f / in_h);
    float shiftX = 0.f ,shiftY = 0.f;
    if(keepration)scaleX = scaleY = scaleX > scaleY ? scaleX : scaleY;
    if(keepration && keepcenter){shiftX = (in_w - w/scaleX)/2.f;shiftY = (in_h - h/scaleY)/2.f;}
    const int n = in_w*in_h;
    const int blockSize = 512;
    const int gridSize = (n + blockSize - 1) / blockSize;

    resizeNormKernel<<<gridSize, blockSize, 0>>>((uchar3*)(p),d,in_w,in_h,w,h,scaleX,scaleY,shiftX,shiftY);
    return 0;
}